#include "hip/hip_runtime.h"
#include <cstddef>
#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <random>
#include <ctime>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <hip/hip_cooperative_groups.h>
#include <cassert>

namespace cg = cooperative_groups;
constexpr unsigned int SHARED_ARRAY_SIZE = 1024;
size_t nSamples = SHARED_ARRAY_SIZE;// << 14; // 2^24
using VALUE_TYPE = unsigned int;
template<typename T>
__device__ inline void swap_if_greater(T& value1, T& value2) {
  if(value1 <= value2) return;
  T t = value1;
  value1 = value2;
  value2 = t;
}

template<typename T>
__global__ void oddEvenBatcherSort(T* data) {
  cg::thread_block cta = cg::this_thread_block();
  __shared__ T shared_buffer[SHARED_ARRAY_SIZE];
  data += blockIdx.x * SHARED_ARRAY_SIZE + threadIdx.x;
  shared_buffer[threadIdx.x] = data[0];
  shared_buffer[threadIdx.x + SHARED_ARRAY_SIZE / 2] = data[SHARED_ARRAY_SIZE / 2];
  for (uint size = 2; size <= SHARED_ARRAY_SIZE; size *= 2) {
    uint stride = size / 2;
    uint offset = threadIdx.x & (stride - 1);

    cg::sync(cta);
    uint pos = 2 * threadIdx.x - offset;
    swap_if_greater(shared_buffer[pos], shared_buffer[pos + stride]);
    stride /= 2;

    for (; stride > 0; stride /= 2) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));

      if (offset >= stride)
        swap_if_greater(shared_buffer[pos - stride], shared_buffer[pos]);
    }
  }

  cg::sync(cta);
  data[0] = shared_buffer[threadIdx.x];
  data[SHARED_ARRAY_SIZE / 2] = shared_buffer[threadIdx.x + SHARED_ARRAY_SIZE / 2];
}
template<typename T>
__global__ void oddEvenBatcherSortMerge(T* data, uint size, uint stride) {
  uint value_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint pos = 2 * value_id - (value_id & (stride - 1));
  if (stride < size / 2) {
    uint offset = value_id & (size / 2 - 1);
    if (offset >= stride)
      swap_if_greater(data[pos - stride], data[pos]);
  } else {
    swap_if_greater(data[pos], data[pos + stride]);
  }
}

void cuda_assert(hipError_t err, const char* text) {
	if(err != hipSuccess) {
		std::cout << text << " (error code " << hipGetErrorString(err)
              << ")!" << std::endl;
		std::exit(EXIT_FAILURE);
	}
}

int main(int argc, const char* argv[]) {
  if(argc > 1) {
    auto shl = std::atoi(argv[1]);
    if(shl > 1 && shl <= 20)
      nSamples <<= shl;
  }
  size_t nBytes = nSamples * sizeof(VALUE_TYPE);

  VALUE_TYPE* host_data = (VALUE_TYPE*)malloc(nSamples * sizeof(VALUE_TYPE));
  std::mt19937_64 generator(time(nullptr));
  std::generate_n(host_data, nSamples, [&generator]() { return(generator()); });
  VALUE_TYPE* device_data = nullptr;
	cuda_assert(hipMalloc((void**)&device_data, nSamples * sizeof(VALUE_TYPE)),
                          "Failed to allocate device memory!");
  cuda_assert(hipMemcpy(device_data, host_data, nBytes,
                          hipMemcpyHostToDevice),
                          "Failed to copy vector from host to device" );
  assert(nSamples >= SHARED_ARRAY_SIZE);
  assert(nSamples % SHARED_ARRAY_SIZE == 0);
  assert((nSamples & (nSamples - 1)) == 0);

  uint blockCount = nSamples / SHARED_ARRAY_SIZE;
  uint threadCount = SHARED_ARRAY_SIZE / 2;

  cuda_assert(hipDeviceSynchronize(), "failed!");
  oddEvenBatcherSort<<<blockCount, threadCount>>>(device_data);
  cuda_assert(hipGetLastError(), "Failed to launch kernel");
  if(blockCount > 1) {
    blockCount = nSamples * 2 / SHARED_ARRAY_SIZE;
    threadCount = SHARED_ARRAY_SIZE / 4;
    for(uint size = 2 * SHARED_ARRAY_SIZE; size <= nSamples; size *= 2) {
      for(unsigned stride = size / 2; stride > 0; stride >>= 1) {
        oddEvenBatcherSortMerge<<<blockCount, threadCount>>>(device_data, size, stride);
        cuda_assert(hipGetLastError(), "Failed to launch kernel");
      }
    }
  }
  
  cuda_assert(hipDeviceSynchronize(), "hipDeviceSynchronize failed!");
	
  cuda_assert(hipMemcpy(host_data, device_data, nBytes, hipMemcpyDeviceToHost),
        "Failed to copy vector from device to host");

  if(std::is_sorted(host_data, host_data + nSamples))
    std::cout << "ok" << std::endl;
  else
    std::cout << "failed" << std::endl;
  
  cuda_assert(hipFree(device_data), "Failed to free device memory");
  free(host_data);

  return(EXIT_SUCCESS);
}
